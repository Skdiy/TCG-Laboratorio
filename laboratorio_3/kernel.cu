#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
using namespace std;

__global__ void MatrixSumKernel(int M, float* Md, float* Nd) {
    // Pvalue es usado para el valor intermedio
    float Pvalue = 0;
    int aux = threadIdx.y * M;
    for (int k = 0; k < M; ++k) {
        Pvalue = Pvalue + Md[aux + k];
    }
    Nd[threadIdx.y] = Pvalue;
}

void SumaColMatrizDevice(float M, float N, float* Mh, float* Nh) {
    int size = M * N * sizeof(float), size2 = N * sizeof(float);
    float* Md, * Nd;
    // Allocate en device
    hipMalloc(&Md, size);
    hipMalloc(&Nd, size2);
    // Inicializo matrices en el device
    hipMemcpy(Md, Mh, size, hipMemcpyHostToDevice);
    hipMemset(Nd, 0, N * sizeof(float));
    // Invocar el kernel que suma en GPU
    dim3 tamGrid(1, 1); //Grid dimensión
    dim3 tamBlock(1, N, 1); //Block dimensión
    // lanzamiento del kernel
    MatrixSumKernel << <tamGrid, tamBlock >> > (M, Md, Nd);
    // Traer resultado;
    hipMemcpy(Nh, Nd, size, hipMemcpyDeviceToHost);
    /*for (int k = 0; k < N; k++)
    {
        cout << "Valor del Vector:" << Nh[k] << endl;
    }*/
    // Free matrices en device
    hipFree(Md); hipFree(Nd);
}

int main()
{
    //fila, columna
    int i, j,a;
    float m = 3;
    float n = 3;
    float* matriz;
    float* vector;
    vector = new float[n];
    matriz = new float[m * n];
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            a = i * n + j;
            matriz[a] = a;
            cout<<"Numero Añadido a la matriz" << matriz[a] << endl;
        }
    }
    for (int l = 0; l < n; l++)
    {
        vector[l] = 0;
        cout << "Vector de Suma" << vector[l] << endl;
    }

    SumaColMatrizDevice(m, n, matriz, vector);
    for (int k = 0; k < n; k++)
    {
        cout <<"Valor del Vector:" << vector[k] << endl;
    }
    return 0;
}