#include "hip/hip_runtime.h"
%%cu
#include <assert.h> 
#include <hip/hip_runtime.h> 
#include <hip/device_functions.h>
#include <>
#include <stdio.h> 

__global__ void SumaColMatrizKernel_0(int M, int N, float* Md, float* Nd){
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = blockIdx.x * blockDim.x + threadIdx.x;
    int posIni = columna * M;
        for (int k = 0; k < M; ++k) {
            Pvalue = Pvalue + Md[posIni + k];
        }
    Nd[columna] = Pvalue;
}
