#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */

// Max Threads per block in GeForce 210
#define TxB 512

__global__
void image_equalization_kernel(const uchar4* const rgbaImage,
                       uchar4* const outputImage,
                       int numRows, int numCols)
{
  // El mapeo de los componentes uchar4 aRGBA es:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //Nota: Ignoramos el canal alfa
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < numRows*numCols){
    uchar4 px = rgbaImage[i]; // thread pixel to process
    //printf( "Antes: R: %i G: %i B %i \n",px.x,px.y,px.z );


    //printf( "Despues: R: %i G: %i B %i \n",outputImage[i].x,outputImage[i].y,outputImage[i].z );
  }
  
}

void image_equalization(uchar4 * const d_rgbaImage,
                  uchar4* const d_outputImage, size_t numRows, size_t numCols)
{

  // Dado que no importa la posicion relativa de los pixels
  // en este algoritmo, la estrategia para asignar hilos a
  // bloques y rejillas sera sencillamente la de cubrir
  // a todos los pixeles con hebras en el eje X
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  image_equalization_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_outputImage, numRows, numCols);
  
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
}